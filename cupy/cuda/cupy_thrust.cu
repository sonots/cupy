#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "cupy_common.h"
#include "cupy_thrust.h"

using namespace thrust;

/*
 * sort
 */

template <typename T>
void cupy::thrust::_sort(void *start, const std::vector<ptrdiff_t>& shape, size_t stream) {

    size_t ndim = shape.size();
    ptrdiff_t size;
    device_ptr<T> dp_first, dp_last;
    hipStream_t _stream = (hipStream_t)stream;

    // Compute the total size of the array.
    size = shape[0];
    for (size_t i = 1; i < ndim; ++i) {
        size *= shape[i];
    }

    dp_first = device_pointer_cast(static_cast<T*>(start));
    dp_last  = device_pointer_cast(static_cast<T*>(start) + size);

    if (ndim == 1) {
        stable_sort(cuda::par.on(_stream), dp_first, dp_last);
    } else {
        device_vector<size_t> d_keys(size);

        // Generate key indices.
        transform(cuda::par.on(_stream),
                  make_counting_iterator<size_t>(0),
                  make_counting_iterator<size_t>(size),
                  make_constant_iterator<ptrdiff_t>(shape[ndim-1]),
                  d_keys.begin(),
                  divides<size_t>());

        // Sorting with back-to-back approach.
        stable_sort_by_key(cuda::par.on(_stream),
                           dp_first,
                           dp_last,
                           d_keys.begin(),
                           less<T>());

        stable_sort_by_key(cuda::par.on(_stream),
                           d_keys.begin(),
                           d_keys.end(),
                           dp_first,
                           less<size_t>());
    }
}

template void cupy::thrust::_sort<cpy_byte>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_ubyte>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_short>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_ushort>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_int>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_uint>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_long>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_ulong>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_float>(void *, const std::vector<ptrdiff_t>&, size_t);
template void cupy::thrust::_sort<cpy_double>(void *, const std::vector<ptrdiff_t>&, size_t);


/*
 * lexsort
 */

template <typename T>
class elem_less {
public:
    elem_less(const T *data):_data(data) {}
    __device__ bool operator()(size_t i, size_t j) { return _data[i] < _data[j]; }
private:
    const T *_data;
};

template <typename T>
void cupy::thrust::_lexsort(size_t *idx_start, void *keys_start, size_t k, size_t n, size_t stream) {
    /* idx_start is the beginning of the output array where the indexes that
       would sort the data will be placed. The original contents of idx_start
       will be destroyed. */
    device_ptr<size_t> dp_first = device_pointer_cast(idx_start);
    device_ptr<size_t> dp_last  = device_pointer_cast(idx_start + n);
    hipStream_t _stream = (hipStream_t)stream;
    sequence(cuda::par.on(_stream), dp_first, dp_last);
    for (size_t i = 0; i < k; ++i) {
        T *key_start = static_cast<T*>(keys_start) + i * n;
        stable_sort< system::cuda::detail::execute_on_stream, device_ptr<size_t> >(
            cuda::par.on(_stream),
            dp_first,
            dp_last,
            elem_less<T>(key_start)
        );
    }
}

template void cupy::thrust::_lexsort<cpy_byte>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_ubyte>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_short>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_ushort>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_int>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_uint>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_long>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_ulong>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_float>(size_t *, void *, size_t, size_t, size_t);
template void cupy::thrust::_lexsort<cpy_double>(size_t *, void *, size_t, size_t, size_t);


/*
 * argsort
 */

template <typename T>
void cupy::thrust::_argsort(size_t *idx_start, void *data_start, size_t num, size_t stream) {
    /* idx_start is the beggining of the output array where the indexes that
       would sort the data will be placed. The original contents of idx_start
       will be destroyed. */

    device_ptr<T> dp_data_first, dp_data_last;
    device_ptr<size_t> dp_idx_first, dp_idx_last;
    hipStream_t _stream = (hipStream_t)stream;

    // Cast device pointers of data.
    dp_data_first = device_pointer_cast(static_cast<T*>(data_start));
    dp_data_last  = device_pointer_cast(static_cast<T*>(data_start) + num);

    // Generate an index sequence.
    dp_idx_first = device_pointer_cast(static_cast<size_t*>(idx_start));
    dp_idx_last  = device_pointer_cast(static_cast<size_t*>(idx_start) + num);
    sequence(cuda::par.on(_stream), dp_idx_first, dp_idx_last);

    // Sort the index sequence by data.
    stable_sort_by_key(cuda::par.on(_stream),
                       dp_data_first,
                       dp_data_last,
                       dp_idx_first,
                       less<T>());
}

template void cupy::thrust::_argsort<cpy_byte>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_ubyte>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_short>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_ushort>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_int>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_uint>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_long>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_ulong>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_float>(size_t *, void *, size_t, size_t);
template void cupy::thrust::_argsort<cpy_double>(size_t *, void *, size_t, size_t);
